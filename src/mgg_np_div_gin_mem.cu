#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <ctime>
#include <algorithm>

#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <hipblas.h>

#include "graph.h"
#include "utils.cuh"
#include "neighbor_utils.cuh"
#include "csr_formatter.h"
#include "layer.h"

#include "cublas_utils.h"
#include "layer_new.cuh"
#include "gnn_layer.cuh"
#include "nccl.h"


// #define validate 1 // the number (< num_GPUs) indicates the validation on which PE.
using nidType = int;
// using nidType = long;

using namespace cudl;
using namespace std;

int main(int argc, char* argv[]){
	
    if (argc < 8){
        printf("Usage: ./main graph.mtx num_GPUs partSize warpPerblock dim interleaved_dist hidden\n");
        return -1;
    }
    
    cout << "Graph File: " << argv[1] << '\n';
    const char *beg_file = argv[1];
	const char *csr_file = argv[2];
	const char *weight_file = argv[3];
    int num_GPUs = atoi(argv[4]);           // 2
    int partSize = atoi(argv[5]);           // 32
    int warpPerBlock = atoi(argv[6]);       // 4
    int interleaved_dist = atoi(argv[7]);   // 2
    int dim = atoi(argv[8]);                // 16
    int hiddenSize = atoi(argv[9]);
    int outdim = atoi(argv[10]);
    float eps = 0.5;

    graph<long, long, nidType, nidType, nidType, nidType>* ginst = new graph<long, long, nidType, nidType, nidType, nidType>(beg_file, csr_file, weight_file);
    std::vector<nidType> global_row_ptr(ginst->beg_pos, ginst->beg_pos + ginst->vert_count + 1);
    std::vector<nidType> global_col_ind(ginst->csr, ginst->csr + ginst->edge_count);

    cout << "Complete loading graphs !!" << endl;
    nidType numNodes = global_row_ptr.size() - 1;
    nidType numEdges = global_col_ind.size();   
    // std::cout << "max node: " << *std::max_element(std::begin(global_col_ind), std::end(global_col_ind)) << '\n';
    
    double t1, t2; 
    // print_array<int>("global_row_ptr", global_row_ptr, global_row_ptr.size());
    // print_array<int>("global_col_ind", global_col_ind, global_col_ind.size());
    int rank, nranks;
    hipStream_t stream;
    nvshmemx_init_attr_t attr;
    MPI_Comm mpi_comm = MPI_COMM_WORLD;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    attr.mpi_comm = &mpi_comm;

    // Set up NVSHMEM device.
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);
    hipStreamCreate(&stream);

    // Set the workload on each device.
    nidType nodesPerPE = (numNodes + num_GPUs - 1) / num_GPUs;
    // printf("numNodes: %d, nodesPerPE: %d\n", numNodes, nodesPerPE);
    nidType lb = nodesPerPE * mype_node;
    nidType ub = (lb + nodesPerPE) < numNodes? (lb + nodesPerPE) : numNodes;
    nidType local_edges = global_row_ptr[ub] - global_row_ptr[lb];
    nidType edge_beg = global_row_ptr[lb];

    std::clock_t c_start_proc = std::clock();    
    // Divide the CSR into the local and remote for each GPU.
    auto split_output = split_CSR<nidType>(global_row_ptr, global_col_ind, lb, ub);
    std::clock_t c_end_proc = std::clock();
    float preproc_time_elapsed_ms = 1000.0 * (c_end_proc - c_start_proc) / CLOCKS_PER_SEC;
    
    if (mype_node == 0)
        printf("Preproc (ms): %.3f\n", preproc_time_elapsed_ms);

    // printf("lb: %d, ub: %d\n", lb, ub);
    auto local_ptr_vec = split_output[0];       // with the base start from lb.
    auto remote_ptr_vec = split_output[1];      // with the base start from ub.
    auto local_col_idx_vec = split_output[2];
    auto remote_col_idx_vec = split_output[3];
    // printf("PE[%d]. local: %d, remote: %d\n", mype_node, local_col_idx_vec.size(), remote_col_idx_vec.size());

    // Allocate memory on each device.
    float *d_input, 
            *dsp_out, 
            *den_out,
            *h_input;

    // d_input = (float *) nvshmem_malloc (nodesPerPE * dim * sizeof(float));  // NVSHMEM global memory for input embedding.
    gpuErrchk(hipMalloc((void**)&d_input, static_cast<size_t>(nodesPerPE) * max(dim, outdim) * sizeof(float))); 
    gpuErrchk(hipMalloc((void**)&dsp_out, static_cast<size_t>(nodesPerPE) * hiddenSize * sizeof(float)));
    // gpuErrchk(hipMalloc((void**)&den_out, nodesPerPE * hiddenSize * sizeof(float))); 
    den_out = (float *) nvshmem_malloc (static_cast<size_t>(nodesPerPE) * hiddenSize * sizeof(float)); 
    h_input = (float *) malloc (nodesPerPE * dim * sizeof(float));          // CPU host memory (input)

    std::fill_n(h_input, nodesPerPE*dim, 1.0f);                             // filled with all ones for input embeddings.
    gpuErrchk(hipMemset(dsp_out, 0, nodesPerPE * hiddenSize * sizeof(float)));

    // d_input (gmem) -dense-> 
    // den_out (nvshmem) -sparse-> dsp_out (gmem) -dense->
    // softmax -> (gmem)
    dense_param_new_beg* dp1 = new dense_param_new_beg("d-1", d_input, den_out, nodesPerPE, dim, hiddenSize);
    dense_param_new_hidden* dp2 = new dense_param_new_hidden("d-2", dsp_out, den_out, nodesPerPE, hiddenSize, hiddenSize);
    dense_param_new_hidden* dp3 = new dense_param_new_hidden("d-3", dsp_out, den_out, nodesPerPE, hiddenSize, hiddenSize);
    dense_param_new_hidden* dp4 = new dense_param_new_hidden("d-4", dsp_out, den_out, nodesPerPE, hiddenSize, hiddenSize);
    dense_param_new_hidden* dp5 = new dense_param_new_hidden("d-5", dsp_out, den_out, nodesPerPE, hiddenSize, hiddenSize);
    dense_param_new_hidden* dp6 = new dense_param_new_hidden("d-6", dsp_out, den_out, nodesPerPE, hiddenSize, outdim);
    softmax_new_param* smx2 = new softmax_new_param("smx-2", den_out, d_input, nodesPerPE, outdim);
 
    #ifdef validate
    float *h_input_ref, *h_output_ref,  *d_input_ref, *d_output_ref;
    if (mype_node == validate)
    {
        h_input_ref = (float *) malloc (numNodes * dim * sizeof(float));      // CPU host memory (input_ref)
        h_output_ref = (float *) malloc (numNodes * dim * sizeof(float));     //  CPU host memory (output_ref)
        std::fill_n(h_input_ref, numNodes * dim, 1.0f); // filled with all zeros.
        std::fill_n(h_output_ref, numNodes * dim, 0.0f); // filled with all zeros.
        gpuErrchk(hipMalloc((void**)&d_input_ref, numNodes * dim * sizeof(float))); // GPU device memory (input_ref)
        gpuErrchk(hipMalloc((void**)&d_output_ref, numNodes * dim * sizeof(float))); // GPU device memory (output_ref)
    }
    #endif

    // graph data structures.
    nidType *d_row_ptr_l, *d_col_ind_l,  *d_row_ptr_r, *d_col_ind_r;
    gpuErrchk(hipMalloc((void**)&d_row_ptr_l, local_ptr_vec.size()*sizeof(nidType))); 
    gpuErrchk(hipMalloc((void**)&d_col_ind_l, local_col_idx_vec.size()*sizeof(nidType))); 
    gpuErrchk(hipMalloc((void**)&d_row_ptr_r, remote_ptr_vec.size()*sizeof(nidType))); 
    gpuErrchk(hipMalloc((void**)&d_col_ind_r, remote_col_idx_vec.size()*sizeof(nidType))); 

    gpuErrchk(hipMemcpy(d_row_ptr_l, &local_ptr_vec[0], local_ptr_vec.size()*sizeof(nidType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_col_ind_l, &local_col_idx_vec[0], local_col_idx_vec.size()*sizeof(nidType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_row_ptr_r, &remote_ptr_vec[0], remote_ptr_vec.size()*sizeof(nidType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_col_ind_r, &remote_col_idx_vec[0], remote_col_idx_vec.size()*sizeof(nidType), hipMemcpyHostToDevice));

    #ifdef validate
    int* d_row_ptr_ref, *d_col_ind_ref;
    if (mype_node == validate)
    {
        gpuErrchk(hipMalloc((void**)&d_row_ptr_ref, global_row_ptr.size()*sizeof(int))); 
        gpuErrchk(hipMalloc((void**)&d_col_ind_ref, global_col_ind.size()*sizeof(int))); 
        gpuErrchk(hipMemcpy(d_row_ptr_ref, &global_row_ptr[0], global_row_ptr.size()*sizeof(int), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_col_ind_ref, &global_col_ind[0], global_col_ind.size()*sizeof(int), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_input_ref, h_input_ref, numNodes * dim * sizeof(float), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_output_ref, h_output_ref, numNodes * dim * sizeof(float), hipMemcpyHostToDevice));
        
        //
        // Compute the result [lb, ub] based on the whole graph CSR.
        //
        SAG_host_ref(d_output_ref, d_input_ref, 
                    d_row_ptr_ref, d_col_ind_ref, 
                    lb, ub, dim);

        gpuErrchk(hipMemcpy(h_output_ref, d_output_ref, numNodes * dim * sizeof(float), hipMemcpyDeviceToHost));
    }
    #endif
    MPI_Barrier(MPI_COMM_WORLD); 

    //
    // Compute on each GPU device.
    //
    // for (int i = 0; i < 10; i++)
    // {
    //     mgg_GIN_np_div(dsp_out, d_input, d_row_ptr_l, d_col_ind_l, d_row_ptr_r, d_col_ind_r,
    //                     lb, ub, dim, nodesPerPE, mype_node, partSize, warpPerBlock, interleaved_dist);
    //     MPI_Barrier(MPI_COMM_WORLD); 
    // }
    
    int num_profiles = 100;
    std::clock_t c_start = std::clock();    
    MPI_Barrier(MPI_COMM_WORLD);
    t1 = MPI_Wtime(); 

    for (int i = 0; i < num_profiles; i++)
    {
        dense_beg_new_forward(dp1);
     
        // layer 1
        mgg_GIN_np_div(dsp_out, dp1->d_out, d_row_ptr_l, d_col_ind_l, d_row_ptr_r, d_col_ind_r,
                        lb, ub, hiddenSize, nodesPerPE, mype_node, partSize, warpPerBlock, interleaved_dist, eps);
        dense_hidden_new_forward(dp2);
       
        //layer 2
        mgg_GIN_np_div(den_out, dp2->d_out, d_row_ptr_l, d_col_ind_l, d_row_ptr_r, d_col_ind_r,
                        lb, ub, hiddenSize, nodesPerPE, mype_node, partSize, warpPerBlock, interleaved_dist, eps);
        dense_hidden_new_forward(dp3);
    
        //layer 3
        mgg_GIN_np_div(den_out, dp3->d_out, d_row_ptr_l, d_col_ind_l, d_row_ptr_r, d_col_ind_r,
        lb, ub, hiddenSize, nodesPerPE, mype_node, partSize, warpPerBlock, interleaved_dist, eps);
        dense_hidden_new_forward(dp4);

        //layer 4
        mgg_GIN_np_div(den_out, dp4->d_out, d_row_ptr_l, d_col_ind_l, d_row_ptr_r, d_col_ind_r,
            lb, ub, hiddenSize, nodesPerPE, mype_node, partSize, warpPerBlock, interleaved_dist, eps);
        dense_hidden_new_forward(dp5);

        //layer 5
        mgg_GIN_np_div(den_out, dp5->d_out, d_row_ptr_l, d_col_ind_l, d_row_ptr_r, d_col_ind_r,
            lb, ub, hiddenSize, nodesPerPE, mype_node, partSize, warpPerBlock, interleaved_dist, eps);
        dense_hidden_new_forward(dp6);

        softmax_new_forward(smx2);
    }

    std::clock_t c_end = std::clock();
    float time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC / num_profiles;
    printf("PE-%d, Total (ms): %.3f\n", mype_node, time_elapsed_ms);
    MPI_Barrier(MPI_COMM_WORLD); 
    t2 = MPI_Wtime(); 
    if (mype_node == 0) printf( "MPI time (ms) %.3f\n", (t2 - t1)*1e3/num_profiles); 
    
    // gpuErrchk(hipMemcpy(h_output, dsp_out, nodesPerPE*dim*sizeof(float), hipMemcpyDeviceToHost));

    #ifdef validate
    if (mype_node == validate){
        for (int nid = 0; nid < 10; nid++){
            printf("out [%d] ", nid);
            for (int d = 0; d < 5; d++){
                printf("%.3f,", h_output[nid * dim + d]);
            }
            printf("\n");
        }
        printf("==============================\n");
        for (int nid = 0; nid < 10; nid++){
            printf("ref [%d] ", nid);
            for (int d = 0; d < 5; d++){
                printf("%.3f,", h_output_ref[lb * dim + nid * dim + d]);
            }
            printf("\n");
        }
        bool val_status = check_equal(h_output_ref, h_output, (ub - lb) * dim, dim, lb * dim);
        printf("Validation on PE-{%d}, status: ", validate);
        if (val_status) printf("True\n"); else printf("False\n");
    }
    #endif

    // release memory.
    hipFree(dsp_out);
    hipFree(d_row_ptr_l);
    hipFree(d_col_ind_l);
    hipFree(d_row_ptr_r);
    hipFree(d_col_ind_r);
    hipFree(d_input);
    hipDeviceReset();

    nvshmem_finalize();

    free(h_input);

    MPI_Finalize();

    #ifdef validate
    if (mype_node == validate){
        hipFree(d_output_ref);
        free(h_output_ref);
    }
    #endif


    if (mype_node == 0) 
        printf("===================================\n");

    return 0;
}
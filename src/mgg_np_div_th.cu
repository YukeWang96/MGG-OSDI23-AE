#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <ctime>
#include <algorithm>

#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <hipblas.h>

#include "graph.h"
#include "utils.cuh"
#include "neighbor_utils.cuh"
#include "csr_formatter.h"
#include "layer.h"

#include "cublas_utils.h"
#include "layer_new.cuh"
#include "gnn_layer.cuh"
#include "nccl.h"


// #define validate 1 // the number (< num_GPUs) indicates the validation on which PE.

using nidType = int;
// using nidType = long;

using namespace cudl;
using namespace std;

int main(int argc, char* argv[]){
	
    if (argc < 8){
        printf("Usage: ./main graph.mtx num_GPUs partSize warpPerblock dim interleaved_dist hidden\n");
        return -1;
    }

    // cout << "Graph File: " << argv[1] << '\n';
	// CSR asym = assemble_csr_matrix_new(argv[1]);
    // auto global_row_ptr = asym.row_ptr;
    // auto global_col_ind = asym.col_ind;

    // const char *beg_file = "dataset/base_0/citeseer_beg_pos.bin";
	// const char *csr_file = "dataset/base_0/citeseer_csr.bin";
	// const char *weight_file = "dataset/base_0/citeseer_weight.bin";
    
    cout << "Graph File: " << argv[1] << '\n';
    const char *beg_file = argv[1];
	const char *csr_file = argv[2];
	const char *weight_file = argv[3];
    
    // const char *beg_file = argv[1];
	// const char *csr_file = "dataset/base_0/citeseer_csr.bin";
	// const char *weight_file = "dataset/base_0/citeseer_weight.bin";
    graph<long, long, nidType, nidType, nidType, nidType>* ginst = new graph<long, long, nidType, nidType, nidType, nidType>(beg_file, csr_file, weight_file);
    std::vector<nidType> global_row_ptr(ginst->beg_pos, ginst->beg_pos + ginst->vert_count + 1);
    std::vector<nidType> global_col_ind(ginst->csr, ginst->csr + ginst->edge_count);

    cout << "Complete loading graphs !!" << endl;
    nidType numNodes = global_row_ptr.size() - 1;
    nidType numEdges = global_col_ind.size();    

    int num_GPUs = atoi(argv[4]);           // 2
    int partSize = atoi(argv[5]);           // 32
    int warpPerBlock = atoi(argv[6]);       // 4
    int dim = atoi(argv[7]);                // 16
    int interleaved_dist = atoi(argv[8]);   // 2
    int hiddenSize = atoi(argv[9]);
    // int outdim = atoi(argv[10]);
    // std::cout << "max node: " << *std::max_element(std::begin(global_col_ind), std::end(global_col_ind)) << '\n';
    
    double t1, t2; 
    // print_array<int>("global_row_ptr", global_row_ptr, global_row_ptr.size());
    // print_array<int>("global_col_ind", global_col_ind, global_col_ind.size());
    int rank, nranks;
    hipStream_t stream;
    nvshmemx_init_attr_t attr;
    MPI_Comm mpi_comm = MPI_COMM_WORLD;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    attr.mpi_comm = &mpi_comm;

    // Set up NVSHMEM device.
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);
    hipStreamCreate(&stream);

    // Set the workload on each device.
    nidType nodesPerPE = (numNodes + num_GPUs - 1) / num_GPUs;
    // printf("numNodes: %d, nodesPerPE: %d\n", numNodes, nodesPerPE);
    nidType lb = nodesPerPE * mype_node;
    nidType ub = (lb + nodesPerPE) < numNodes? (lb + nodesPerPE) : numNodes;
    nidType local_edges = global_row_ptr[ub] - global_row_ptr[lb];
    nidType edge_beg = global_row_ptr[lb];

    std::clock_t c_start_proc = std::clock();    
    // Divide the CSR into the local and remote for each GPU.
    auto split_output = split_CSR<nidType>(global_row_ptr, global_col_ind, lb, ub);
    std::clock_t c_end_proc = std::clock();
    float preproc_time_elapsed_ms = 1000.0 * (c_end_proc - c_start_proc) / CLOCKS_PER_SEC;
    if (mype_node == 0)
    printf("Preproc (ms): %.3f\n", preproc_time_elapsed_ms);

    // printf("lb: %d, ub: %d\n", lb, ub);
    auto local_ptr_vec = split_output[0];       // with the base start from lb.
    auto remote_ptr_vec = split_output[1];      // with the base start from ub.
    auto local_col_idx_vec = split_output[2];
    auto remote_col_idx_vec = split_output[3];
    // printf("PE[%d]. local: %d, remote: %d\n", mype_node, local_col_idx_vec.size(), remote_col_idx_vec.size());
    // MPI_Finalize();
    // exit(0);

    // Allocate memory on each device.
    float *d_input, *d_output, *h_input, *h_output;
    gpuErrchk(hipMalloc((void**)&d_output, nodesPerPE * dim * sizeof(float))); 
    d_input = (float *) nvshmem_malloc (nodesPerPE * dim * sizeof(float));  // NVSHMEM global memory for input embedding.
    h_input = (float *) malloc (nodesPerPE * dim * sizeof(float));          // CPU host memory (input)
    h_output = (float *) malloc (nodesPerPE * dim * sizeof(float));         //  CPU host memory (output)
    std::fill_n(h_input, nodesPerPE*dim, 1.0f); // filled with all ones for input embeddings.
    std::fill_n(h_output, nodesPerPE*dim, 0.0f); // filled with all zeros for output embeddings.
    gpuErrchk(hipMemcpy(d_input, h_input, nodesPerPE * dim * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_output, h_output, nodesPerPE * dim * sizeof(float), hipMemcpyHostToDevice));

    // dense_param_beg* dp1 = new dense_param_beg("d-1", numNodes, dim, dim1);
    dense_param_hidden* dp2 = new dense_param_hidden("d-2", d_output, nodesPerPE, dim, dim);
    softmax_param* smx2 = new softmax_param("smx-2", dp2->d_out, nodesPerPE, dim);

    #ifdef validate
    float *h_input_ref, *h_output_ref,  *d_input_ref, *d_output_ref;
    if (mype_node == validate)
    {
        h_input_ref = (float *) malloc (numNodes * dim * sizeof(float));      // CPU host memory (input_ref)
        h_output_ref = (float *) malloc (numNodes * dim * sizeof(float));     //  CPU host memory (output_ref)
        std::fill_n(h_input_ref, numNodes * dim, 1.0f); // filled with all zeros.
        std::fill_n(h_output_ref, numNodes * dim, 0.0f); // filled with all zeros.
        gpuErrchk(hipMalloc((void**)&d_input_ref, numNodes * dim * sizeof(float))); // GPU device memory (input_ref)
        gpuErrchk(hipMalloc((void**)&d_output_ref, numNodes * dim * sizeof(float))); // GPU device memory (output_ref)
    }
    #endif

    nidType *d_row_ptr_l, *d_col_ind_l,  *d_row_ptr_r, *d_col_ind_r;
    gpuErrchk(hipMalloc((void**)&d_row_ptr_l, local_ptr_vec.size()*sizeof(nidType))); 
    gpuErrchk(hipMalloc((void**)&d_col_ind_l, local_col_idx_vec.size()*sizeof(nidType))); 
    gpuErrchk(hipMalloc((void**)&d_row_ptr_r, remote_ptr_vec.size()*sizeof(nidType))); 
    gpuErrchk(hipMalloc((void**)&d_col_ind_r, remote_col_idx_vec.size()*sizeof(nidType))); 

    gpuErrchk(hipMemcpy(d_row_ptr_l, &local_ptr_vec[0], local_ptr_vec.size()*sizeof(nidType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_col_ind_l, &local_col_idx_vec[0], local_col_idx_vec.size()*sizeof(nidType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_row_ptr_r, &remote_ptr_vec[0], remote_ptr_vec.size()*sizeof(nidType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_col_ind_r, &remote_col_idx_vec[0], remote_col_idx_vec.size()*sizeof(nidType), hipMemcpyHostToDevice));

    #ifdef validate
    int* d_row_ptr_ref, *d_col_ind_ref;
    if (mype_node == validate)
    {
        gpuErrchk(hipMalloc((void**)&d_row_ptr_ref, global_row_ptr.size()*sizeof(int))); 
        gpuErrchk(hipMalloc((void**)&d_col_ind_ref, global_col_ind.size()*sizeof(int))); 
        gpuErrchk(hipMemcpy(d_row_ptr_ref, &global_row_ptr[0], global_row_ptr.size()*sizeof(int), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_col_ind_ref, &global_col_ind[0], global_col_ind.size()*sizeof(int), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_input_ref, h_input_ref, numNodes * dim * sizeof(float), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_output_ref, h_output_ref, numNodes * dim * sizeof(float), hipMemcpyHostToDevice));
        
        //
        // Compute the result [lb, ub] based on the whole graph CSR.
        //
        SAG_host_ref(d_output_ref, d_input_ref, 
                    d_row_ptr_ref, d_col_ind_ref, 
                    lb, ub, dim);

        gpuErrchk(hipMemcpy(h_output_ref, d_output_ref, numNodes * dim * sizeof(float), hipMemcpyDeviceToHost));
    }
    #endif
    MPI_Barrier(MPI_COMM_WORLD); 

    //
    // Compute on each GPU device.
    //
    for (int i = 0; i < 10; i++)
    {
        mgg_SAG_np_div_th(d_output, d_input, d_row_ptr_l, d_col_ind_l, d_row_ptr_r, d_col_ind_r,
                        lb, ub, dim, nodesPerPE, mype_node, partSize, warpPerBlock, interleaved_dist);
        MPI_Barrier(MPI_COMM_WORLD); 
    }
    
    int num_profiles = 100;
    std::clock_t c_start = std::clock();    
    MPI_Barrier(MPI_COMM_WORLD);
    t1 = MPI_Wtime(); 

    for (int i = 0; i < num_profiles; i++)
    {
        mgg_SAG_np_div_th(d_output, d_input, d_row_ptr_l, d_col_ind_l, d_row_ptr_r, d_col_ind_r,
                        lb, ub, dim, nodesPerPE, mype_node, partSize, warpPerBlock, interleaved_dist);
        MPI_Barrier(MPI_COMM_WORLD); 

        // dense_hidden_forward(dp2);
        // MPI_Barrier(MPI_COMM_WORLD); 
        // nvshmem_float_sum_reduce(NVSHMEMX_TEAM_NODE, dp2->d_W_new, dp2->d_W, dp2->dim1*dp2->dim2);

        // mgg_SAG_np_div(d_output, d_input, d_row_ptr_l, d_col_ind_l, d_row_ptr_r, d_col_ind_r,
        //                 lb, ub, dim, nodesPerPE, mype_node, partSize, warpPerBlock, interleaved_dist);
        // MPI_Barrier(MPI_COMM_WORLD); 

        // dense_hidden_forward(dp2);
        // MPI_Barrier(MPI_COMM_WORLD); 
        // nvshmem_float_sum_reduce(NVSHMEMX_TEAM_NODE, dp2->d_W_new, dp2->d_W, dp2->dim1*dp2->dim2);

        // softmax_forward(smx2);
    }

    std::clock_t c_end = std::clock();
    float time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC / num_profiles;
    printf("PE-%d, Total (ms): %.3f\n", mype_node, time_elapsed_ms);
    MPI_Barrier(MPI_COMM_WORLD); 
    t2 = MPI_Wtime(); 
    if (mype_node == 0) printf( "MPI time (ms) %.3f\n", (t2 - t1)*1e3/num_profiles); 
    
    gpuErrchk(hipMemcpy(h_output, d_output, nodesPerPE*dim*sizeof(float), hipMemcpyDeviceToHost));

    #ifdef validate
    if (mype_node == validate){
        for (int nid = 0; nid < 10; nid++){
            printf("out [%d] ", nid);
            for (int d = 0; d < 5; d++){
                printf("%.3f,", h_output[nid * dim + d]);
            }
            printf("\n");
        }
        printf("==============================\n");
        for (int nid = 0; nid < 10; nid++){
            printf("ref [%d] ", nid);
            for (int d = 0; d < 5; d++){
                printf("%.3f,", h_output_ref[lb * dim + nid * dim + d]);
            }
            printf("\n");
        }
        bool val_status = check_equal(h_output_ref, h_output, (ub - lb) * dim, dim, lb * dim);
        printf("Validation on PE-{%d}, status: ", validate);
        if (val_status) printf("True\n"); else printf("False\n");
    }
    #endif

    // release memory.
    hipFree(d_output);
    hipFree(d_row_ptr_l);
    hipFree(d_col_ind_l);
    hipFree(d_row_ptr_r);
    hipFree(d_col_ind_r);
    nvshmem_free(d_input);
    nvshmem_finalize();

    free(h_input);
    free(h_output);

    MPI_Finalize();

    #ifdef validate
    if (mype_node == validate){
        hipFree(d_output_ref);
        free(h_output_ref);
    }
    #endif


    if (mype_node == 0) 
        printf("===================================\n");

    return 0;
}
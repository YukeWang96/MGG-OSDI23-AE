#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <ctime>
#include <algorithm>

#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <hipblas.h>

#include "utils.cuh"
#include "neighbor_utils.cuh"
#include "csr_formatter.h"
#include "layer.h"

// #define validate 0 // the number (< num_GPUs) indicates the validation on which PE.

using namespace cudl;
using namespace std;

int main(int argc, char* argv[]){
	
    if (argc < 8){
        printf("Usage: ./main graph.mtx num_GPUs partSize warpPerblock dim interleaved_dist hidden\n");
        return -1;
    }

    cout << "Graph File: " << argv[1] << '\n';
	CSR asym = assemble_csr_matrix_new(argv[1]);
    cout << "Complete loading graphs !!" << endl;

    int numNodes = asym.row_ptr.size() - 1;
    // std::cout << "max node: " << *std::max_element(std::begin(asym.col_ind), std::end(asym.col_ind)) << '\n';
    int numEdges = asym.col_ind.size();    
    int num_GPUs = atoi(argv[2]);           // 2
    int partSize = atoi(argv[3]);           // 32
    int warpPerBlock = atoi(argv[4]);       // 4
    int dim = atoi(argv[5]);                // 16
    int interleaved_dist = atoi(argv[6]);   // 2
    int hiddenSize = atoi(argv[7]);

    double t1, t2; 
    // print_array<int>("asym.row_ptr", asym.row_ptr, asym.row_ptr.size());
    // print_array<int>("asym.col_ind", asym.col_ind, asym.col_ind.size());
    int rank, nranks;
    hipStream_t stream;
    nvshmemx_init_attr_t attr;
    MPI_Comm mpi_comm = MPI_COMM_WORLD;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    attr.mpi_comm = &mpi_comm;

    // Set up NVSHMEM device.
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);
    hipStreamCreate(&stream);

    // Set the workload on each device.
    int nodesPerPE = (numNodes + num_GPUs - 1) / num_GPUs;
    printf("numNodes: %d, nodesPerPE: %d\n", numNodes, nodesPerPE);
    int lb = nodesPerPE * mype_node;
    int ub = (lb + nodesPerPE) < numNodes? (lb + nodesPerPE) : numNodes;
    int local_nodes = ub - lb;
    int local_edges = asym.row_ptr[ub] - asym.row_ptr[lb];
    int edge_beg = asym.row_ptr[lb];

    // Allocate memory on each device.
    float *d_input, *d_output, *h_input, *h_output;
    gpuErrchk(hipMalloc((void**)&d_output, nodesPerPE * dim * sizeof(float))); 
    d_input = (float *) nvshmem_malloc (nodesPerPE * dim * sizeof(float)); // NVSHMEM global memory
    h_input = (float *) malloc (nodesPerPE * dim * sizeof(float));      // CPU host memory (input)
    h_output = (float *) malloc (nodesPerPE * dim * sizeof(float));     //  CPU host memory (output)
    std::fill_n(h_input, nodesPerPE*dim, 1.0f); // filled with all ones.
    std::fill_n(h_output, nodesPerPE*dim, 0.0f); // filled with all zeros.

    #ifdef validate
    float *h_input_ref, *h_output_ref,  *d_input_ref, *d_output_ref;
    if (mype_node == validate)
    {
        h_input_ref = (float *) malloc (numNodes * dim * sizeof(float));      // CPU host memory (input_ref)
        h_output_ref = (float *) malloc (numNodes * dim * sizeof(float));     //  CPU host memory (output_ref)
        std::fill_n(h_input_ref, numNodes * dim, 1.0f); // filled with all zeros.
        std::fill_n(h_output_ref, numNodes * dim, 0.0f); // filled with all zeros.
        gpuErrchk(hipMalloc((void**)&d_input_ref, numNodes * dim * sizeof(float))); // GPU device memory (input_ref)
        gpuErrchk(hipMalloc((void**)&d_output_ref, numNodes * dim * sizeof(float))); // GPU device memory (output_ref)
    }
    #endif

    int *d_row_ptr, *d_col_ind;
    gpuErrchk(hipMalloc((void**)&d_row_ptr, (local_nodes + 1)*sizeof(int))); 
    gpuErrchk(hipMalloc((void**)&d_col_ind, local_edges*sizeof(int))); 
    gpuErrchk(hipMemcpy(d_row_ptr, &asym.row_ptr[lb], (local_nodes + 1)*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_col_ind, &asym.col_ind[edge_beg], local_edges*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_input, h_input, nodesPerPE*dim*sizeof(float), hipMemcpyHostToDevice));

    #ifdef validate
    int* d_row_ptr_ref, *d_col_ind_ref;
    if (mype_node == validate)
    {
        gpuErrchk(hipMalloc((void**)&d_row_ptr_ref, asym.row_ptr.size()*sizeof(int))); 
        gpuErrchk(hipMalloc((void**)&d_col_ind_ref, asym.col_ind.size()*sizeof(int))); 
        gpuErrchk(hipMemcpy(d_row_ptr_ref, &asym.row_ptr[0], asym.row_ptr.size()*sizeof(int), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_col_ind_ref, &asym.col_ind[0], asym.col_ind.size()*sizeof(int), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_input_ref, h_input_ref, numNodes * dim * sizeof(float), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_output_ref, h_output_ref, numNodes * dim * sizeof(float), hipMemcpyHostToDevice));

        SAG_host_ref(d_output_ref, d_input_ref, 
                    d_row_ptr_ref, d_col_ind_ref, 
                    lb, ub, dim);

        gpuErrchk(hipMemcpy(h_output_ref, d_output_ref, numNodes * dim * sizeof(float), hipMemcpyDeviceToHost));
    }
    #endif
    MPI_Barrier(MPI_COMM_WORLD); 

    //
    // Compute on each device.
    //
    std::clock_t c_start = std::clock();    
    MPI_Barrier(MPI_COMM_WORLD);
    t1 = MPI_Wtime(); 

    mgg_SAG_basic(d_output, d_input, d_row_ptr, d_col_ind,
                    lb, ub, dim, nodesPerPE);

    gpuErrchk(hipGetLastError());
    gpuErrchk(hipDeviceSynchronize());

    std::clock_t c_end = std::clock();
    float time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    printf("PE-%d, Total (ms): %.3f\n", mype_node, time_elapsed_ms);
    MPI_Barrier(MPI_COMM_WORLD); 
    t2 = MPI_Wtime(); 
    if (mype_node == 0) printf( "MPI time (ms) %.3f\n", (t2 - t1)*1e3); 
    
    gpuErrchk(hipMemcpy(h_output, d_output, nodesPerPE*dim*sizeof(float), hipMemcpyDeviceToHost));


    #ifdef validate
    if (mype_node == validate){
        for (int nid = 0; nid < 10; nid++){
            printf("out [%d] ", nid);
            for (int d = 0; d < 5; d++){
                printf("%.3f,", h_output[nid * dim + d]);
            }
            printf("\n");
        }
        printf("==============================\n");
        for (int nid = 0; nid < 10; nid++){
            printf("ref [%d] ", nid);
            for (int d = 0; d < 5; d++){
                printf("%.3f,", h_output_ref[lb * dim + nid * dim + d]);
            }
            printf("\n");
        }
        bool val_status = check_equal(h_output_ref, h_output, (ub - lb) * dim, lb * dim);
        printf("Validation on PE-{%d}, status: ", validate);
        if (val_status) printf("True\n"); else printf("False\n");
    }
    #endif

    // release memory.
    hipFree(d_output);
    hipFree(d_row_ptr);
    hipFree(d_col_ind);

    nvshmem_free(d_input);
    nvshmem_finalize();
    MPI_Finalize();
    
    if (mype_node == 0) 
        printf("===================================\n");

    return 0;
}
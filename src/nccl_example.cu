#include <stdio.h>
#include <ctime>
#include "hip/hip_runtime.h"
#include "nccl.h"

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


int main(int argc, char* argv[])
{
  if (argc < 3){
    printf("./exe ndevices size\n");
    exit(-1);
  }
  //managing 4 devices
  int nDev = atoi(argv[1]);
  int size = atoi(argv[2]);
  int *devs = new int[nDev];
  ncclComm_t* comms = (ncclComm_t*)malloc(sizeof(ncclComm_t)*nDev);
  // ncclComm_t comms[4];

  for (int i = 0; i < nDev; i++)
      devs[i] = i;

  //allocating and initializing device buffers
  float** sendbuff = (float**)malloc(nDev * sizeof(float*));
  float** recvbuff = (float**)malloc(nDev * sizeof(float*));
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);


  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
    CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
    CUDACHECK(hipMemset(sendbuff[i], 1, size * sizeof(float)));
    CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(float)));
    CUDACHECK(hipStreamCreate(s+i));
  }

  // cudaEvent_t start, stop;
  // cudaEventCreate(&start);
  // cudaEventCreate(&stop);
  // cudaEventRecord(start);
  
  std::clock_t dense_start = std::clock();

  //initializing NCCL
  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

  //calling NCCL communication API. Group API is required when using
  //multiple devices per thread
  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < nDev; ++i){
    ncclSend(sendbuff[i], size, ncclFloat, (i+1)%nDev, comms[i], s[i]);
    ncclRecv(recvbuff[i], size, ncclFloat, (i-1+nDev)%nDev, comms[i], s[i]);
    // NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], 
    //                         size, ncclFloat, ncclSum, comms[i], s[i]));
  }
  NCCLCHECK(ncclGroupEnd());


  std::clock_t dense_end = std::clock();
  float dense_time_elapsed_ms = 1000.0 * (dense_end - dense_start) / CLOCKS_PER_SEC;
  printf("CPU-Wall (ms): %.3f\n", dense_time_elapsed_ms);

  // cudaEventRecord(stop);
  // cudaEventSynchronize(stop);
  // float milliseconds = 0;
  // cudaEventElapsedTime(&milliseconds, start, stop);
  // printf("kernel time (ms): %.3f\n", milliseconds);

  //synchronizing on CUDA streams to wait for completion of NCCL operation
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }

  //free device buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
  }

  //finalizing NCCL
  for(int i = 0; i < nDev; ++i)
      ncclCommDestroy(comms[i]);

  printf("Success \n");
  return 0;
}
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


int main(int argc, char* argv[])
{
  if (argc < 3){
    printf("./exe ndevices dim\n");
    exit(-1);
  }
  //managing 4 devices
  // int nDev = 3;
  int nDev = atoi(argv[1]);
  // int size = 32*1024*1024;
  int size = atoi(argv[2]);
  int *devs = new int[nDev];
  ncclComm_t* comms = (ncclComm_t*)malloc(sizeof(ncclComm_t)*nDev);
  // ncclComm_t comms[4];

  for (int i = 0; i < nDev; i++)
      devs[i] = i;

  //allocating and initializing device buffers
  float** sendbuff = (float**)malloc(nDev * sizeof(float*));
  float** recvbuff = (float**)malloc(nDev * sizeof(float*));
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);


  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
    CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
    CUDACHECK(hipMemset(sendbuff[i], 1, size * sizeof(float)));
    CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(float)));
    CUDACHECK(hipStreamCreate(s+i));
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  
  //initializing NCCL
  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

   //calling NCCL communication API. Group API is required when using
   //multiple devices per thread
  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < nDev; ++i){
    ncclSend(sendbuff[i], size, ncclFloat, (i+1)%nDev, comms[i], s[i]);
    ncclRecv(recvbuff[i], size, ncclFloat, (i-1+nDev)%nDev, comms[i], s[i]);
    // NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], 
    //                         size, ncclFloat, ncclSum, comms[i], s[i]));
  }
  NCCLCHECK(ncclGroupEnd());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("kernel time (ms): %.3f\n", milliseconds);


  //synchronizing on CUDA streams to wait for completion of NCCL operation
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }

  //free device buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
  }

  //finalizing NCCL
  for(int i = 0; i < nDev; ++i)
      ncclCommDestroy(comms[i]);

  printf("Success \n");
  return 0;
}
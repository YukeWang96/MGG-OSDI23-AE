#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <ctime>
#include <algorithm>

#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <hipblas.h>

#include "graph.h"
#include "utils.cuh"
#include "neighbor_utils.cuh"
#include "csr_formatter.h"
#include "layer.h"

#include "cublas_utils.h"
#include "layer_new.cuh"
#include "gnn_layer.cuh"
#include "nccl.h"


// #define validate 1 // the number (< num_GPUs) indicates the validation on which PE.
// using nidType = size_t;
// using nidType = long;
using nidType = int;

using namespace cudl;
using namespace std;

int main(int argc, char* argv[]){
	
    if (argc < 8){
        printf("Usage: ./main graph.mtx num_GPUs partSize warpPerblock dim interleaved_dist hidden\n");
        return -1;
    }
    
    cout << "Graph File: " << argv[1] << '\n';
    const char *beg_file = argv[1];
	const char *csr_file = argv[2];
	const char *weight_file = argv[3];
    int num_GPUs = atoi(argv[4]);           // 2
    int partSize = atoi(argv[5]);           // 32
    int warpPerBlock = atoi(argv[6]);       // 4
    int interleaved_dist = atoi(argv[7]);   // 2
    int dim = atoi(argv[8]);                // 16
    int hiddenSize = atoi(argv[9]);
    int outdim = atoi(argv[10]);

    int max_dim = max({hiddenSize, outdim});
    int max_dim_1 = max({dim, outdim});

    graph<long, long, nidType, nidType, nidType, nidType>* ginst = new graph<long, long, nidType, nidType, nidType, nidType>(beg_file, csr_file, weight_file);
    std::vector<nidType> global_row_ptr(ginst->beg_pos, ginst->beg_pos + ginst->vert_count + 1);
    std::vector<nidType> global_col_ind(ginst->csr, ginst->csr + ginst->edge_count);

    cout << "Complete loading graphs !!" << endl;
    nidType numNodes = global_row_ptr.size() - 1;
    nidType numEdges = global_col_ind.size();    


    // std::cout << "max node: " << *std::max_element(std::begin(global_col_ind), std::end(global_col_ind)) << '\n';
    
    double t1, t2; 
    // print_array<int>("global_row_ptr", global_row_ptr, global_row_ptr.size());
    // print_array<int>("global_col_ind", global_col_ind, global_col_ind.size());
    int rank, nranks;
    hipStream_t stream;
    nvshmemx_init_attr_t attr;
    MPI_Comm mpi_comm = MPI_COMM_WORLD;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    attr.mpi_comm = &mpi_comm;

    // Set up NVSHMEM device.
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);
    hipStreamCreate(&stream);

    // Set the workload on each device.
    nidType nodesPerPE = (numNodes + num_GPUs - 1) / num_GPUs;
    // printf("numNodes: %d, nodesPerPE: %d\n", numNodes, nodesPerPE);
    nidType lb = nodesPerPE * mype_node;
    nidType ub = (lb + nodesPerPE) < numNodes? (lb + nodesPerPE) : numNodes;
    nidType local_edges = global_row_ptr[ub] - global_row_ptr[lb];
    nidType edge_beg = global_row_ptr[lb];

    std::clock_t c_start_proc = std::clock();    
    // Divide the CSR into the local and remote for each GPU.
    auto split_output = split_CSR<nidType>(global_row_ptr, global_col_ind, lb, ub);
    std::clock_t c_end_proc = std::clock();
    float preproc_time_elapsed_ms = 1000.0 * (c_end_proc - c_start_proc) / CLOCKS_PER_SEC;
    if (mype_node == 0)
    printf("Preproc (ms): %.3f\n", preproc_time_elapsed_ms);

    // printf("lb: %d, ub: %d\n", lb, ub);
    auto local_ptr_vec = split_output[0];       // with the base start from lb.
    auto remote_ptr_vec = split_output[1];      // with the base start from ub.
    auto local_col_idx_vec = split_output[2];
    auto remote_col_idx_vec = split_output[3];
    // printf("PE[%d]. local: %d, remote: %d\n", mype_node, local_col_idx_vec.size(), remote_col_idx_vec.size());

    // Allocate memory on each device.
    float *h_input;
    float *d_buff_1, *d_buff_2, *d_buff_nvshmem;

    // d_input = (float *) nvshmem_malloc (nodesPerPE * dim * sizeof(float));  // NVSHMEM global memory for input embedding.
    /*
    gpuErrchk(hipMalloc((void**)&d_input, nodesPerPE * dim * sizeof(float))); 
    gpuErrchk(hipMalloc((void**)&dsp_out, nodesPerPE * hiddenSize * sizeof(float))); 
    gpuErrchk(hipMalloc((void**)&dsp_out_1, nodesPerPE * outdim * sizeof(float))); 
    */
    

    // buffers for switching between input and output for each layer.
    printf("d_buff_1: %.3f GB\n", (nodesPerPE * 1.0f * max_dim_1 * sizeof(float))/1e9);
    printf("nodesPerPE: %d, dim: %d\n", nodesPerPE, dim);
    gpuErrchk(hipMalloc((void**)&d_buff_1, static_cast<size_t>(nodesPerPE) * max_dim_1 * sizeof(float))); 
    gpuErrchk(hipMalloc((void**)&d_buff_2, static_cast<size_t>(nodesPerPE) * hiddenSize * sizeof(float))); 
   
    h_input = (float *) malloc (static_cast<size_t>(nodesPerPE) * max_dim_1 * sizeof(float));                  // CPU host memory (input)
    // h_output = (float *) malloc (nodesPerPE * hiddenSize * sizeof(float));         //  CPU host memory (output)
    // hsp_output_1 = (float *) malloc (nodesPerPE * outdim * sizeof(float));         //  CPU host memory (output)
    std::fill_n(h_input, static_cast<size_t>(nodesPerPE)*dim, 1.0f);                                 // filled with all ones for input embeddings.
    // std::fill_n(h_output, nodesPerPE*hiddenSize, 0.0f);                         // filled with all zeros for output embeddings.
    
     /*
    gpuErrchk(hipMemcpy(d_input, h_input, nodesPerPE * dim * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dsp_out, h_output, nodesPerPE * hiddenSize * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dsp_out_1, hsp_output_1, nodesPerPE * outdim * sizeof(float), hipMemcpyHostToDevice));
    */

    gpuErrchk(hipMemcpy(d_buff_1, h_input, static_cast<size_t>(nodesPerPE) * dim * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemset(d_buff_2, 0, static_cast<size_t>(nodesPerPE) * hiddenSize * sizeof(float)));
    d_buff_nvshmem = (float *) nvshmem_malloc (static_cast<size_t>(nodesPerPE) * hiddenSize * sizeof(float)); 

    // Initialize the parameters. 
    // d_buf_1 - (dense-1) -> d_buff_nvshmem -> (sp-1) 
    // -> d_buff_2 -> (dense-2) -> d_buff_nvshmem - (sp-2) -> d_buff_2 
    // -> smx-2 -> d_buff_2
    dense_param_new_beg* dp1 = new dense_param_new_beg("d-1", d_buff_1, d_buff_nvshmem, nodesPerPE, dim, hiddenSize);
    dense_param_new_hidden* dp2 = new dense_param_new_hidden("d-2", d_buff_2, d_buff_nvshmem, nodesPerPE, hiddenSize, outdim);
    softmax_new_param* smx2 = new softmax_new_param("smx-2", d_buff_1, d_buff_1, nodesPerPE, outdim);
 
    #ifdef validate
    float *h_input_ref, *h_output_ref,  *d_input_ref, *d_output_ref;
    if (mype_node == validate)
    {
        h_input_ref = (float *) malloc (numNodes * dim * sizeof(float));      // CPU host memory (input_ref)
        h_output_ref = (float *) malloc (numNodes * dim * sizeof(float));     //  CPU host memory (output_ref)
        std::fill_n(h_input_ref, numNodes * dim, 1.0f); // filled with all zeros.
        std::fill_n(h_output_ref, numNodes * dim, 0.0f); // filled with all zeros.
        gpuErrchk(hipMalloc((void**)&d_input_ref, numNodes * dim * sizeof(float))); // GPU device memory (input_ref)
        gpuErrchk(hipMalloc((void**)&d_output_ref, numNodes * dim * sizeof(float))); // GPU device memory (output_ref)
    }
    #endif

    nidType *d_row_ptr_l, *d_col_ind_l,  *d_row_ptr_r, *d_col_ind_r;
    gpuErrchk(hipMalloc((void**)&d_row_ptr_l, local_ptr_vec.size()*sizeof(nidType))); 
    gpuErrchk(hipMalloc((void**)&d_col_ind_l, local_col_idx_vec.size()*sizeof(nidType))); 
    gpuErrchk(hipMalloc((void**)&d_row_ptr_r, remote_ptr_vec.size()*sizeof(nidType))); 
    gpuErrchk(hipMalloc((void**)&d_col_ind_r, remote_col_idx_vec.size()*sizeof(nidType))); 

    gpuErrchk(hipMemcpy(d_row_ptr_l, &local_ptr_vec[0], local_ptr_vec.size()*sizeof(nidType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_col_ind_l, &local_col_idx_vec[0], local_col_idx_vec.size()*sizeof(nidType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_row_ptr_r, &remote_ptr_vec[0], remote_ptr_vec.size()*sizeof(nidType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_col_ind_r, &remote_col_idx_vec[0], remote_col_idx_vec.size()*sizeof(nidType), hipMemcpyHostToDevice));

    #ifdef validate
    int* d_row_ptr_ref, *d_col_ind_ref;
    if (mype_node == validate)
    {
        gpuErrchk(hipMalloc((void**)&d_row_ptr_ref, global_row_ptr.size()*sizeof(int))); 
        gpuErrchk(hipMalloc((void**)&d_col_ind_ref, global_col_ind.size()*sizeof(int))); 
        gpuErrchk(hipMemcpy(d_row_ptr_ref, &global_row_ptr[0], global_row_ptr.size()*sizeof(int), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_col_ind_ref, &global_col_ind[0], global_col_ind.size()*sizeof(int), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_input_ref, h_input_ref, numNodes * dim * sizeof(float), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_output_ref, h_output_ref, numNodes * dim * sizeof(float), hipMemcpyHostToDevice));
        
        //
        // Compute the result [lb, ub] based on the whole graph CSR.
        //
        SAG_host_ref(d_output_ref, d_input_ref, 
                    d_row_ptr_ref, d_col_ind_ref, 
                    lb, ub, dim);

        gpuErrchk(hipMemcpy(h_output_ref, d_output_ref, numNodes * dim * sizeof(float), hipMemcpyDeviceToHost));
    }
    #endif
    MPI_Barrier(MPI_COMM_WORLD); 

    //
    // Compute on each GPU device.
    //
    // for (int i = 0; i < 10; i++)
    // {
    //     mgg_SAG_np_div(dsp_out, d_input, d_row_ptr_l, d_col_ind_l, d_row_ptr_r, d_col_ind_r,
    //                     lb, ub, dim, nodesPerPE, mype_node, partSize, warpPerBlock, interleaved_dist);
    //     MPI_Barrier(MPI_COMM_WORLD); 
    // }
    
    int num_profiles = 100;
    std::clock_t c_start = std::clock();    
    MPI_Barrier(MPI_COMM_WORLD);
    t1 = MPI_Wtime(); 

    for (int i = 0; i < num_profiles; i++)
    {
        dense_beg_new_forward(dp1);
        mgg_SAG_np_div(d_buff_2, dp1->d_out, d_row_ptr_l, d_col_ind_l, d_row_ptr_r, d_col_ind_r,
                        lb, ub, hiddenSize, nodesPerPE, mype_node, partSize, warpPerBlock, interleaved_dist);
        // MPI_Barrier(MPI_COMM_WORLD); 
        // MPI_Barrier(MPI_COMM_WORLD); 
        // nvshmem_float_sum_reduce(NVSHMEMX_TEAM_NODE, dp2->d_W_new, dp2->d_W, dp2->dim1*dp2->dim2);

        dense_hidden_new_forward(dp2);
        mgg_SAG_np_div(d_buff_1, dp2->d_out, d_row_ptr_l, d_col_ind_l, d_row_ptr_r, d_col_ind_r,
                        lb, ub, hiddenSize, nodesPerPE, mype_node, partSize, warpPerBlock, interleaved_dist);
        // MPI_Barrier(MPI_COMM_WORLD); 
        // MPI_Barrier(MPI_COMM_WORLD); 
        // nvshmem_float_sum_reduce(NVSHMEMX_TEAM_NODE, dp2->d_W_new, dp2->d_W, dp2->dim1*dp2->dim2);
        softmax_new_forward(smx2);
    }

    std::clock_t c_end = std::clock();
    float time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC / num_profiles;
    printf("PE-%d, Total (ms): %.3f\n", mype_node, time_elapsed_ms);
    MPI_Barrier(MPI_COMM_WORLD); 
    t2 = MPI_Wtime(); 
    if (mype_node == 0) printf( "MPI time (ms) %.3f\n", (t2 - t1)*1e3/num_profiles); 
    
    // gpuErrchk(hipMemcpy(h_output, dsp_out, nodesPerPE*dim*sizeof(float), hipMemcpyDeviceToHost));

    #ifdef validate
    if (mype_node == validate){
        for (int nid = 0; nid < 10; nid++){
            printf("out [%d] ", nid);
            for (int d = 0; d < 5; d++){
                printf("%.3f,", h_output[nid * dim + d]);
            }
            printf("\n");
        }
        printf("==============================\n");
        for (int nid = 0; nid < 10; nid++){
            printf("ref [%d] ", nid);
            for (int d = 0; d < 5; d++){
                printf("%.3f,", h_output_ref[lb * dim + nid * dim + d]);
            }
            printf("\n");
        }
        bool val_status = check_equal(h_output_ref, h_output, (ub - lb) * dim, dim, lb * dim);
        printf("Validation on PE-{%d}, status: ", validate);
        if (val_status) printf("True\n"); else printf("False\n");
    }
    #endif

    // release memory.
    // hipFree(dsp_out);
    hipFree(d_row_ptr_l);
    hipFree(d_col_ind_l);
    hipFree(d_row_ptr_r);
    hipFree(d_col_ind_r);
    // hipFree(d_input);
    // hipDeviceReset();
    nvshmem_finalize();

    // free(h_input);
    // free(h_output);

    MPI_Finalize();

    #ifdef validate
    if (mype_node == validate){
        hipFree(d_output_ref);
        free(h_output_ref);
    }
    #endif


    if (mype_node == 0) 
        printf("===================================\n");

    return 0;
}
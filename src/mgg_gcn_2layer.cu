#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <ctime>
#include <algorithm>

#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <hipblas.h>

#include "graph.h"
#include "utils.cuh"
#include "neighbor_utils.cuh"
#include "cublas_utils.h"
// #include "csr_formatter.h"
// #include "layer.h"

// using namespace cudl;
using namespace std;

int main(int argc, char* argv[]){
	
    if (argc < 8){
        printf("Usage: ./main graph.mtx num_GPUs partSize warpPerblock dim\n");
        return -1;
    }
    
    cout << "Graph File: " << argv[1] << '\n';
    const char *beg_file = argv[1];
	const char *csr_file = argv[2];
	const char *weight_file = argv[3];

    graph<long, long, int, int, int, int>* ginst = new graph<long, long, int, int, int, int>(beg_file, csr_file, weight_file);
    std::vector<int> global_row_ptr(ginst->beg_pos, ginst->beg_pos + ginst->vert_count + 1);
    std::vector<int> global_col_ind(ginst->csr, ginst->csr + ginst->edge_count);
    // cout << "Complete loading graphs !!" << endl;
    int numNodes = global_row_ptr.size() - 1;
    int numEdges = global_col_ind.size();    

    int num_GPUs = atoi(argv[4]);           // 2
    int partSize = atoi(argv[5]);           // 32
    int warpPerBlock = atoi(argv[6]);       // 4
    int dim = atoi(argv[7]);                // 16
    int dim1 = 128;               
    int dim2 = 128;

    int lb = 0;
    int ub = numNodes;
    
    float* d_row_ptr, *d_col_ind;


    CUDA_CHECK(hipMalloc((void**)&d_row_ptr, global_row_ptr.size()*sizeof(int))); 
    CUDA_CHECK(hipMalloc((void**)&d_col_ind, global_col_ind.size()*sizeof(int))); 
    CUDA_CHECK(hipMemcpy(d_row_ptr, &global_row_ptr[0], global_row_ptr.size()*sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_col_ind, &global_col_ind[0], global_col_ind.size()*sizeof(int), hipMemcpyHostToDevice));



    sparse_param_beg* sp1 = new sparse_param_beg(d_row_ptr, d_col_ind, numNodes, dim);
    dense_param_hidden* dp1 = new dense_param_hidden(sp1->d_out, numNodes, dim, dim1);
    sparse_param_hidden* sp2 = new sparse_param_hidden(dp1->d_out, d_row_ptr, d_col_ind, numNodes, dim1);
    dense_param_hidden* dp1 = new dense_param_hidden(sp2->d_out, numNodes, dim1, dim2);

    //
    // xecute model.
    //
    std::clock_t c_start = std::clock();    
    // sparse layer-1
    SAG_host_ref(sp1->d_out, sp1->d_in, d_row_ptr, d_col_ind, lb, ub, dim1, global_col_ind.size());
    // dense layer-1
    CUBLAS_CHECK(hipblasSgemm(dp1->cublasH, dp1->transa, dp1->transb, dp1->m, dp1->n, dp1->k, &(dp1->malpha), dp1->d_W, dp1->ldw, dp1->d_out, dp1->ldx, &(dp1->beta), dp1->d_out, dp1->ldout));
    // sparse layer-2
    SAG_host_ref(sp2->d_out, sp2->d_in, d_row_ptr, d_col_ind, lb, ub, dim1, global_col_ind.size());
    // dense layer-2
    CUBLAS_CHECK(hipblasSgemm(dp2->cublasH, dp2->transa, dp2->transb, dp2->m, dp2->n, dp2->k, &(dp2->malpha), dp2->d_W, dp2->ldw, dp2->d_out, dp2->ldx, &(dp2->beta), dp2->d_out, dp2->ldout));

    std::clock_t c_end = std::clock();
    float time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    printf("Total (ms): %.3f\n", time_elapsed_ms);


    // CUDA_CHECK(hipMemcpy(h_output_ref, d_output_ref, numNodes * dim * sizeof(float), hipMemcpyDeviceToHost));
    // CUDA_CHECK(hipMemcpyAsync(C.data(), d_C, sizeof(data_type) * C.size(), hipMemcpyDeviceToHost, stream));  
    //
    // sparse-1 param
    // 
    // float *h_input_ref, *h_output_ref,  *d_input_ref, *d_output_ref;
    // h_input_ref = (float *) malloc (numNodes * dim * sizeof(float));      // CPU host memory (input_ref)
    // h_output_ref = (float *) malloc (numNodes * dim * sizeof(float));     //  CPU host memory (output_ref)
    // std::fill_n(h_input_ref, numNodes * dim, 1.0f); // filled with all zeros.
    // std::fill_n(h_output_ref, numNodes * dim, 0.0f); // filled with all zeros.
    // CUDA_CHECK(hipMalloc((void**)&d_input_ref, numNodes * dim * sizeof(float))); // GPU device memory (input_ref)
    // CUDA_CHECK(hipMalloc((void**)&d_output_ref, numNodes * dim * sizeof(float))); // GPU device memory (output_ref)

    // int* d_row_ptr, *d_col_ind;
    // CUDA_CHECK(hipMalloc((void**)&d_row_ptr, global_row_ptr.size()*sizeof(int))); 
    // CUDA_CHECK(hipMalloc((void**)&d_col_ind, global_col_ind.size()*sizeof(int))); 
    // CUDA_CHECK(hipMemcpy(d_row_ptr, &global_row_ptr[0], global_row_ptr.size()*sizeof(int), hipMemcpyHostToDevice));
    // CUDA_CHECK(hipMemcpy(d_col_ind, &global_col_ind[0], global_col_ind.size()*sizeof(int), hipMemcpyHostToDevice));
    // CUDA_CHECK(hipMemcpy(d_input_ref, h_input_ref, numNodes * dim * sizeof(float), hipMemcpyHostToDevice));
    // CUDA_CHECK(hipMemcpy(d_output_ref, h_output_ref, numNodes * dim * sizeof(float), hipMemcpyHostToDevice));
    
    //
    // dense-1 layer param
    //
    // const float alpha = 1.0f;
    // const float beta = 0.0;

    // hipblasOperation_t transa = HIPBLAS_OP_N;
    // hipblasOperation_t transb = HIPBLAS_OP_N;
    // hipblasHandle_t cublasH = NULL;
    // hipStream_t stream = NULL;

    // CUBLAS_CHECK(hipblasCreate(&cublasH));
    // CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    // CUBLAS_CHECK(hipblasSetStream(cublasH, stream));
    
    // float* h_W1, *d_W1, *d_out1, *d_out1_sag;
    // h_W1 = (float *) malloc (dim * dim1 * sizeof(float));      // CPU host memory (input_ref)
    // std::fill_n(h_W1, dim * dim1, 1.0f); // filled with all zeros.
    // CUDA_CHECK(hipMalloc((void**)&d_W1, dim * dim1 * sizeof(float))); // GPU device memory (input_ref)
    // CUDA_CHECK(hipMemcpy(d_W1, h_W1, dim * dim1 * sizeof(float), hipMemcpyHostToDevice));
    // CUDA_CHECK(hipMalloc((void**)&d_out1, numNodes * dim1 * sizeof(float)));
    // CUDA_CHECK(hipMalloc((void**)&d_out1_sag, numNodes * dim1 * sizeof(float)));

    // const int m1 = dim1, n1 = numNodes, k1 = dim; // (XW) --> W_T x X_T for column-major store.
    // const int ldx1 = dim, ldw1 = dim1, ldout1 = dim1;
    //
    // dense-2 layer param
    //    
    // float* h_W2, *d_W2, *d_out2;
    // h_W2 = (float *) malloc (dim1 * dim2 * sizeof(float));      // CPU host memory (input_ref)
    // std::fill_n(h_W1, dim1 * dim2, 1.0f); // filled with all zeros.
    // CUDA_CHECK(hipMalloc((void**)&d_W2,  dim1 * dim2 * sizeof(float))); // GPU device memory (input_ref)
    // CUDA_CHECK(hipMemcpy(d_W2, h_W2,  dim1 * dim2 * sizeof(float), hipMemcpyHostToDevice));
    // CUDA_CHECK(hipMalloc((void**)&d_out2, numNodes * dim2 * sizeof(float)));

    // const int m2 = dim2, n2 = numNodes, k2 = dim1; // (XW) --> W_T X X_T for column-major store.
    // const int ldx2 = dim1, ldw2 = dim2, ldout2 = dim2;

    // for (int nid = 0; nid < 10; nid++){
    //     printf("out [%d] ", nid);
    //     for (int d = 0; d < dim; d++){
    //         printf("%.3f,", h_output_ref[nid * dim + d]);
    //     }
    //     printf("\n");
    // }

    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess){
    //     printf("CUDA error @ SAG_cuda_kernel_ref: %s\n", hipGetErrorString(error));
    //     exit(-1);
    // }


    hipFree(d_output_ref);
    free(h_output_ref);
    printf("===================================\n");

    return 0;
}

#include <iostream>
#include <stdio.h>
#include <omp.h>

#include "utils.cuh"
#include "neighbor_utils.cuh"
#include "csr_formatter.h"

using namespace std;


int main(int argc, char* argv[]){
	
    if (argc < 4){
        printf("Usage: ./main graph.mtx dim nodeOfInterest\n");
        return -1;
    }

    cout << "Graph File: " << argv[1] << '\n';
	CSR asym = assemble_csr_matrix_new(argv[1]);
    cout << "Complete loading graphs !!" << endl;

    int numNodes = asym.row_ptr.size() - 1;
    int numEdges = asym.col_ind.size();

    int warpPerBlock = 1;
    int dim = atoi(argv[2]);

    // float* input = (float*)malloc(numNodes*dim*sizeof(float));
    float *d_output, *d_input;
    int *d_col_ind, *d_row_ptr;

    int mype_node = 0;
    hipSetDevice(mype_node);

    // Load the corresponding tiles.
    const int lb_src = 0; // node of interest
    const int ub_src = lb_src + 1;  // the node next to the node of interest.
    const int e_lb = 0;
    const int e_ub = e_lb + atoi(argv[3]);
    printf("node [%d]: %d neighbors\n", lb_src, asym.row_ptr[ub_src] - asym.row_ptr[lb_src]);
    
    gpuErrchk(hipMallocManaged((void**)&d_input, numNodes*dim*sizeof(float)));    // UVM allocation
    // gpuErrchk(hipMalloc((void**)&d_output, (ub_src-lb_src)*dim*sizeof(float)));   
    gpuErrchk(hipMalloc((void**)&d_output, (ub_src-lb_src)*dim*sizeof(float)));   

    gpuErrchk(hipMalloc((void**)&d_row_ptr, numNodes*sizeof(int))); 
    gpuErrchk(hipMalloc((void**)&d_col_ind, numEdges*sizeof(int))); 

    gpuErrchk(hipMemcpy(d_row_ptr, &asym.row_ptr[0], numNodes*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_col_ind, &asym.col_ind[0], numEdges*sizeof(int), hipMemcpyHostToDevice));


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    
    uvm_profile<<<1, 32*warpPerBlock>>>(d_output, d_input, 
                                        d_row_ptr, d_col_ind, 
                                        e_lb, e_ub, dim);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        printf("[%d] CUDA error at uvm_profile: %s\n", mype_node, hipGetErrorString(error));
        exit(-1);
    }
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time (ms): %.2f\n", milliseconds);
    printf("===================================\n");

    hipFree(d_input);    
    hipFree(d_output);
    hipFree(d_col_ind);

    return 0;
}
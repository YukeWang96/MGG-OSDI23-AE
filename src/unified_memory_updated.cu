
#include <iostream>
#include <stdio.h>
#include <omp.h>

#include "graph.h"
#include "utils.cuh"
#include "neighbor_utils.cuh"
#include "csr_formatter.h"

// #define validate 0 //--> for results validation

using namespace std;

int main(int argc, char* argv[]){
	
    if (argc < 5){
        printf("Usage: ./main beg_file.bin csr_file.bin weight_file.bin num_GPUs partSize warpPerBlock dim\n");
        return -1;
    }

    cout << "Graph File: " << argv[1] << '\n';
    const char *beg_file = argv[1];
	const char *csr_file = argv[2];
	const char *weight_file = argv[3];
    
    graph<long, long, int, int, int, int>* ginst = new graph<long, long, int, int, int, int>(beg_file, csr_file, weight_file);
    std::vector<int> global_row_ptr(ginst->beg_pos, ginst->beg_pos + ginst->vert_count + 1);
    std::vector<int> global_col_ind(ginst->csr, ginst->csr + ginst->edge_count);

    int numNodes = global_row_ptr.size() - 1;
    int numEdges = global_col_ind.size();    

    int num_GPUs = atoi(argv[4]);
    int partSize = atoi(argv[5]);
    int warpPerBlock = atoi(argv[6]);
    int dim = atoi(argv[7]);

    int nodesPerPE = (numNodes + num_GPUs - 1) / num_GPUs;
    float** h_input = new float*[num_GPUs];
    float** h_output = new float*[num_GPUs];
    int **d_row_ptr = new int*[num_GPUs]; 
    int **d_col_ind = new int*[num_GPUs]; 

    float **d_input;
    gpuErrchk(hipMallocManaged((void**)&d_input,  num_GPUs*sizeof(float*))); 

// #ifdef validate
//     float* h_ref = (float*)malloc(nodesPerPE*dim*sizeof(float));
//     float *d_ref;
//     gpuErrchk(hipMallocManaged((void**)&d_ref,   nodesPerPE*dim*sizeof(float))); // input: device 2D pointer
//     std::fill(h_ref, h_ref+nodesPerPE*dim, 0.0);          // sets every value in the array to 0.0
//     hipMemset(d_ref, 0, nodesPerPE*dim*sizeof(float));
// #endif

#pragma omp parallel for
for (int mype_node = 0; mype_node < num_GPUs; mype_node++)
{
    hipSetDevice(mype_node);

    h_input[mype_node] = (float*)malloc(nodesPerPE*dim*sizeof(float));
    h_output[mype_node] = (float*)malloc(nodesPerPE*dim*sizeof(float));

    std::fill(h_input[mype_node], h_input[mype_node]+nodesPerPE*dim, 1.0);      // sets every value in the array to 1.0
    std::fill(h_output[mype_node], h_output[mype_node]+nodesPerPE*dim, 0.0);    // sets every value in the array to 0.0

    printf("mype_node: %d, nodesPerPE: %d\n", mype_node, nodesPerPE);

    // UVM data: output, input, row_ptr, col_ind 
    gpuErrchk(hipMallocManaged((void**)&d_input[mype_node],   nodesPerPE*dim*sizeof(float))); // input: device 2D pointer
    gpuErrchk(hipMallocManaged((void**)&h_output[mype_node],  nodesPerPE*dim*sizeof(float))); // output: host pointer
    gpuErrchk(hipMallocManaged((void**)&d_row_ptr[mype_node], (numNodes+1)*sizeof(int)));
    gpuErrchk(hipMallocManaged((void**)&d_col_ind[mype_node], numEdges*sizeof(int))); 

    gpuErrchk(hipMemcpy(d_input[mype_node],   h_input[mype_node],  nodesPerPE*dim*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_row_ptr[mype_node], &global_row_ptr[0],  (numNodes+1)*sizeof(int),     hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_col_ind[mype_node], &global_col_ind[0],  numEdges*sizeof(int),         hipMemcpyHostToDevice));
}


// #ifdef validate
//     hipSetDevice(validate);
//     SAG_host_single_ref(d_ref,      d_input[validate],  d_row_ptr[validate], d_col_ind[validate], numNodes, dim);
//     gpuErrchk(hipMemcpy(h_ref,     d_ref,              nodesPerPE*dim*sizeof(float),   hipMemcpyDeviceToHost));
// #endif

// One GPU per threads
#pragma omp parallel for
for (int mype_node = 0; mype_node < num_GPUs; mype_node++)
{
    hipSetDevice(mype_node);

    const int lb_src = nodesPerPE * mype_node;
    const int ub_src = min_val(lb_src+nodesPerPE, numNodes);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    SAG_host_UVM_updated(h_output[mype_node], d_input, 
                        d_row_ptr[mype_node], d_col_ind[mype_node], 
                        lb_src, ub_src, dim, num_GPUs, 
                        mype_node, nodesPerPE);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time (ms): %.2f\n", milliseconds);
}

// #pragma omp parallel for
// for (int mype_node = 0; mype_node < num_GPUs; mype_node++)
// {
    // gpuErrchk(hipMemcpy(h_output[mype_node],  h_output[mype_node], nodesPerPE*dim*sizeof(float), hipMemcpyDeviceToHost));
    // #ifdef validate
    // bool status = compare_array(h_ref, h_output[mype_node], nodesPerPE*dim);
    // printf(status ? "validate: True\n" : "validate: False\n");
    // #endif

    // hipFree(d_ref);    
    // hipFree(d_input[mype_node]);    
    // hipFree(d_output[mype_node]);
    // hipFree(d_col_ind[mype_node]);
    // hipFree(d_row_ptr[mype_node]);
    // hipFree(d_input);
// }

    // free(d_output);
    // free(d_col_ind);
    // free(d_row_ptr);

    // free(h_ref);
    // free(h_output);
    // free(h_input);

    return 0;
}
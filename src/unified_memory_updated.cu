
#include <iostream>
#include <stdio.h>
#include <omp.h>

#include "graph.h"
#include "utils.cuh"
#include "neighbor_utils.cuh"
#include "csr_formatter.h"

// #define validate //--> for results validation
using namespace std;

int main(int argc, char* argv[]){
	
    if (argc < 5){
        printf("Usage: ./main beg_file.bin csr_file.bin weight_file.bin num_GPUs partSize warpPerBlock dim\n");
        return -1;
    }

    cout << "Graph File: " << argv[1] << '\n';
    const char *beg_file = argv[1];
	const char *csr_file = argv[2];
	const char *weight_file = argv[3];
    
    graph<long, long, int, int, int, int>* ginst = new graph<long, long, int, int, int, int>(beg_file, csr_file, weight_file);
    std::vector<int> global_row_ptr(ginst->beg_pos, ginst->beg_pos + ginst->vert_count + 1);
    std::vector<int> global_col_ind(ginst->csr, ginst->csr + ginst->edge_count);

    int numNodes = global_row_ptr.size() - 1;
    int numEdges = global_col_ind.size();    

    int num_GPUs = atoi(argv[4]);
    int partSize = atoi(argv[5]);
    int warpPerBlock = atoi(argv[6]);
    int dim = atoi(argv[7]);

    int nodesPerPE = (numNodes + num_GPUs - 1) / num_GPUs;
    float** h_input = new float*[num_GPUs];
    float** h_output = new float*[num_GPUs];
    float **d_output, **d_input;
    // std::fill(h_ref, h_ref+nodesPerPE*dim, 0.0);          // sets every value in the array to 0.0

    // memset(input, 0, numNodes*dim*sizeof(float));
    int **d_row_ptr = new int*[num_GPUs]; 
    int **d_col_ind = new int*[num_GPUs]; 
    // int *d_row_ptr, *d_col_ind;
    // float *d_ref;


#pragma omp parallel for
for (int mype_node = 0; mype_node < num_GPUs; mype_node++)
{
    hipSetDevice(mype_node);

    // h_input[mype_node] = (float*)malloc(nodesPerPE*dim*sizeof(float));
    // h_output[mype_node] = (float*)malloc(nodesPerPE*dim*sizeof(float));
    // float* h_ref = (float*)malloc(nodesPerPE*dim*sizeof(float));

    // std::fill(h_input[mype_node], h_input[mype_node]+nodesPerPE*dim, 1.0);      // sets every value in the array to 1.0
    // std::fill(h_output[mype_node], h_output[mype_node]+nodesPerPE*dim, 0.0);    // sets every value in the array to 0.0

    printf("mype_node: %d, nodesPerPE: %d\n", mype_node, nodesPerPE);

    gpuErrchk(hipMalloc((void**)&d_input,  num_GPUs*sizeof(float*))); 
    gpuErrchk(hipMalloc((void**)&d_output,  num_GPUs*sizeof(float*))); 
    // UVM data: output, input, row_ptr, col_ind 
    gpuErrchk(hipMallocManaged((void**)&h_output[mype_node],  nodesPerPE*dim*sizeof(float))); 
    gpuErrchk(hipMallocManaged((void**)&h_input[mype_node],   nodesPerPE*dim*sizeof(float))); 
    gpuErrchk(hipMallocManaged((void**)&d_row_ptr[mype_node], (numNodes+1)*sizeof(int)));
    gpuErrchk(hipMallocManaged((void**)&d_col_ind[mype_node], numEdges*sizeof(int))); 

    // hipMemset(d_output[mype_node],            0,         nodesPerPE*dim*sizeof(float));
    // gpuErrchk(hipMemcpy(d_input[mype_node],   h_input[mype_node],   nodesPerPE*dim*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_row_ptr[mype_node], &global_row_ptr[0],   (numNodes+1)*sizeof(int),     hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_col_ind[mype_node], &global_col_ind[0],   numEdges*sizeof(int),         hipMemcpyHostToDevice));


}


// #ifdef validate
// hipSetDevice(0);
// SAG_host_single_ref(d_ref, d_input, d_row_ptr, d_col_ind, numNodes, dim);
// gpuErrchk(hipMemcpy(h_ref,     d_ref,       numNodes*dim*sizeof(float),   hipMemcpyDeviceToHost));
// #endif

// One GPU per threads
#pragma omp parallel for
for (int mype_node = 0; mype_node < num_GPUs; mype_node++)
{
    hipSetDevice(mype_node);
    gpuErrchk(hipMemcpy(d_input, h_input, num_GPUs*sizeof(float*), hipMemcpyHostToDevice));
    // gpuErrchk(hipMemcpy(d_output, h_output, num_GPUs*sizeof(float*), hipMemcpyHostToDevice));

    const int lb_src = nodesPerPE * mype_node;
    const int ub_src = min_val(lb_src+nodesPerPE, numNodes);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    SAG_host_UVM_updated(h_output[mype_node], d_input, 
                        d_row_ptr[mype_node], d_col_ind[mype_node], 
                        lb_src, ub_src, dim, num_GPUs, 
                        mype_node, nodesPerPE);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time (ms): %.2f\n", milliseconds);
}
return;

#pragma omp parallel for
for (int mype_node = 0; mype_node < num_GPUs; mype_node++)
{
    gpuErrchk(hipMemcpy(h_output[mype_node],  d_output[mype_node], nodesPerPE*dim*sizeof(float), hipMemcpyDeviceToHost));

    // #ifdef validate
    // bool status = compare_array(h_ref, h_output, numNodes*dim);
    // printf(status ? "validate: True\n" : "validate: False\n");
    // #endif

    // hipFree(d_ref);    
    // hipFree(d_input[mype_node]);    
    // hipFree(d_output[mype_node]);
    // hipFree(d_col_ind[mype_node]);
    // hipFree(d_row_ptr[mype_node]);
    // hipFree(d_input);
}


    // free(d_output);
    // free(d_col_ind);
    // free(d_row_ptr);

    // free(h_ref);
    // free(h_output);
    // free(h_input);

    return 0;
}
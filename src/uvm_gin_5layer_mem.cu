#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <omp.h>

#include "graph.h"
#include "utils.cuh"
#include "neighbor_utils.cuh"
#include "csr_formatter.h"


#include "cublas_utils.h"
#include "layer_new.cuh"
#include "gnn_layer.cuh"

// #define validate 1 //--> for results validation

using namespace std;
// using nidType = int;
// using nidType = long;

int main(int argc, char* argv[]){
	
    if (argc < 5){
        printf("Usage: ./main beg_file.bin csr_file.bin weight_file.bin num_GPUs partSize warpPerBlock dim\n");
        return -1;
    }

    cout << "Graph File: " << argv[1] << '\n';
    const char *beg_file = argv[1];
	const char *csr_file = argv[2];
	const char *weight_file = argv[3];
    
    int num_GPUs = atoi(argv[4]);
    int partSize = atoi(argv[5]);
    int warpPerBlock = atoi(argv[6]);
    
    int dim = atoi(argv[7]);
    int hiddenSize = atoi(argv[8]);
    int outdim = atoi(argv[9]);

    graph<long, long, nidType, nidType, nidType, nidType>* ginst = new graph<long, long, nidType, nidType, nidType, nidType>(beg_file, csr_file, weight_file);
    std::vector<nidType> global_row_ptr(ginst->beg_pos, ginst->beg_pos + ginst->vert_count + 1);
    std::vector<nidType> global_col_ind(ginst->csr, ginst->csr + ginst->edge_count);

    nidType numNodes = global_row_ptr.size() - 1;
    nidType numEdges = global_col_ind.size();    

    int nodesPerPE = (numNodes + num_GPUs - 1) / num_GPUs;
    float** h_input = new float*[num_GPUs];
    // float** d_den_out = new float*[num_GPUs];
    // float** d_den_out_2 = new float*[num_GPUs];

    nidType **d_row_ptr = new nidType*[num_GPUs]; 
    nidType **d_col_ind = new nidType*[num_GPUs]; 

    float   **d_input, 
            **d_den_out; 
            // **d_den_out_2; 
            // **d_den_out_3, 
            // **d_den_out_4, 
            // **d_den_out_5, 
            // **d_den_out_6;
            
    gpuErrchk(hipMallocManaged((void**)&d_input,       num_GPUs*sizeof(float*))); 
    gpuErrchk(hipMallocManaged((void**)&d_den_out,     num_GPUs*sizeof(float*))); 
    // gpuErrchk(hipMallocManaged((void**)&d_den_out_2,   num_GPUs*sizeof(float*))); 
    // gpuErrchk(hipMallocManaged((void**)&d_den_out_3,   num_GPUs*sizeof(float*))); 
    // gpuErrchk(hipMallocManaged((void**)&d_den_out_4,   num_GPUs*sizeof(float*))); 
    // gpuErrchk(hipMallocManaged((void**)&d_den_out_5,   num_GPUs*sizeof(float*))); 
    // gpuErrchk(hipMallocManaged((void**)&d_den_out_6,   num_GPUs*sizeof(float*))); 

#ifdef validate
    float *hd_ref, *hd_input_ref;
    gpuErrchk(hipMallocManaged((void**)&hd_ref,         nodesPerPE*dim*sizeof(float)));   // output reference
    gpuErrchk(hipMallocManaged((void**)&hd_input_ref,   numNodes*dim*sizeof(float)));   // input reference.

    std::fill(hd_input_ref, hd_input_ref + numNodes*dim, 1.0);                           
    std::fill(hd_ref, hd_ref + nodesPerPE*dim, 0.0);                                  
#endif

#pragma omp parallel for
for (int mype_node = 0; mype_node < num_GPUs; mype_node++)
{
    hipSetDevice(mype_node);

    h_input[mype_node] = (float*)malloc(nodesPerPE*dim*sizeof(float));
    std::fill(h_input[mype_node], h_input[mype_node]+nodesPerPE*dim, 1.0);      // sets every value in the array to 1.0
    printf("mype_node: %d, nodesPerPE: %d\n", mype_node, nodesPerPE);

    // global pointers array on each device
    gpuErrchk(hipMallocManaged((void**)&d_input[mype_node],   nodesPerPE*max({dim,outdim}) *sizeof(float))); // input: device 2D pointer
    gpuErrchk(hipMallocManaged((void**)&d_den_out[mype_node], nodesPerPE*hiddenSize*sizeof(nidType)));
    // gpuErrchk(hipMallocManaged((void**)&d_den_out_2[mype_node], nodesPerPE*hiddenSize*sizeof(nidType)));
    // gpuErrchk(hipMallocManaged((void**)&d_den_out_3[mype_node], nodesPerPE*hiddenSize*sizeof(nidType)));
    // gpuErrchk(hipMallocManaged((void**)&d_den_out_4[mype_node], nodesPerPE*hiddenSize*sizeof(nidType)));
    // gpuErrchk(hipMallocManaged((void**)&d_den_out_5[mype_node], nodesPerPE*hiddenSize*sizeof(nidType)));
    // gpuErrchk(hipMallocManaged((void**)&d_den_out_6[mype_node], nodesPerPE*outdim*sizeof(nidType)));

    gpuErrchk(hipMallocManaged((void**)&d_row_ptr[mype_node], (numNodes+1)*sizeof(nidType)));
    gpuErrchk(hipMallocManaged((void**)&d_col_ind[mype_node], numEdges*sizeof(nidType))); 

    gpuErrchk(hipMemcpy(d_input[mype_node],   h_input[mype_node],  nodesPerPE*dim*sizeof(float),   hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_row_ptr[mype_node], &global_row_ptr[0],  (numNodes+1)*sizeof(nidType),   hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_col_ind[mype_node], &global_col_ind[0],  numEdges*sizeof(nidType),       hipMemcpyHostToDevice));
}


#ifdef validate
    hipSetDevice(validate);
    int lb_src_val = nodesPerPE * validate;
    int ub_src_val = min_val(lb_src_val+nodesPerPE, numNodes);
    SAG_UVM_ref(hd_ref, hd_input_ref,  d_row_ptr[validate], d_col_ind[validate], ub_src_val, lb_src_val, numNodes, dim);
#endif

// One GPU per threads
#pragma omp parallel for
for (int mype_node = 0; mype_node < num_GPUs; mype_node++)
{
    hipSetDevice(mype_node);

    float *dsp_out;

    gpuErrchk(hipMalloc((void**)&dsp_out,   nodesPerPE*hiddenSize*sizeof(float))); // output: device pointer
    // gpuErrchk(hipMalloc((void**)&dsp_out_1, nodesPerPE*outdim*sizeof(float))); // output: device pointer
    // gpuErrchk(hipMalloc((void**)&dsp_out_2, nodesPerPE*outdim*sizeof(float))); // output: device pointer
    // gpuErrchk(hipMalloc((void**)&dsp_out_3, nodesPerPE*outdim*sizeof(float))); // output: device pointer
    // gpuErrchk(hipMalloc((void**)&dsp_out_4, nodesPerPE*outdim*sizeof(float))); // output: device pointer
    
    gpuErrchk(hipMemset(dsp_out, 0, nodesPerPE * hiddenSize * sizeof(float)));
    // gpuErrchk(hipMemset(dsp_out_1, 0, nodesPerPE * hiddenSize * sizeof(float)));
    // gpuErrchk(hipMemset(dsp_out_2, 0, nodesPerPE * hiddenSize * sizeof(float)));
    // gpuErrchk(hipMemset(dsp_out_3, 0, nodesPerPE * hiddenSize * sizeof(float)));
    // gpuErrchk(hipMemset(dsp_out_4, 0, nodesPerPE * hiddenSize * sizeof(float)));

    dense_param_beg_uvm* dp1 = new dense_param_beg_uvm("d-1", d_input[mype_node], mype_node, d_den_out, nodesPerPE, dim, hiddenSize);
    dense_param_hidden_uvm* dp2 = new dense_param_hidden_uvm("d-2", dsp_out, mype_node, d_den_out, nodesPerPE, hiddenSize, hiddenSize);
    dense_param_hidden_uvm* dp3 = new dense_param_hidden_uvm("d-3", dsp_out, mype_node, d_den_out, nodesPerPE, hiddenSize, hiddenSize);
    dense_param_hidden_uvm* dp4 = new dense_param_hidden_uvm("d-4", dsp_out, mype_node, d_den_out, nodesPerPE, hiddenSize, hiddenSize);
    dense_param_hidden_uvm* dp5 = new dense_param_hidden_uvm("d-5", dsp_out, mype_node, d_den_out, nodesPerPE, hiddenSize, hiddenSize);
    dense_param_hidden_uvm* dp6 = new dense_param_hidden_uvm("d-6", dsp_out, mype_node, d_den_out, nodesPerPE, hiddenSize, outdim);
    // softmax_new_param* smx2 = new softmax_new_param("smx-2", d_input[mype_node], nodesPerPE, outdim);

    const int lb_src = nodesPerPE * mype_node;
    const int ub_src = min_val(lb_src+nodesPerPE, numNodes);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    //layer-0
    dense_beg_forward_uvm(dp1);

    //layer-1
    SAG_host_UVM_updated(dsp_out, d_den_out, 
                        d_row_ptr[mype_node], d_col_ind[mype_node], 
                        lb_src, ub_src, hiddenSize, num_GPUs, 
                        mype_node, nodesPerPE, numNodes);
    dense_hidden_forward_uvm(dp2);

    // //layer-2
    // SAG_host_UVM_updated(dsp_out_1, d_den_out_2, 
    //                     d_row_ptr[mype_node], d_col_ind[mype_node],
    //                     lb_src, ub_src, outdim, num_GPUs,
    //                     mype_node, nodesPerPE, numNodes);
    // dense_hidden_forward_uvm(dp3);

    // //layer-3
    // SAG_host_UVM_updated(dsp_out_2, d_den_out_3, 
    //                     d_row_ptr[mype_node], d_col_ind[mype_node],
    //                     lb_src, ub_src, outdim, num_GPUs,
    //                     mype_node, nodesPerPE, numNodes);
    // dense_hidden_forward_uvm(dp4);

    // //layer-4
    // SAG_host_UVM_updated(dsp_out_3, d_den_out_4, 
    //                     d_row_ptr[mype_node], d_col_ind[mype_node],
    //                     lb_src, ub_src, outdim, num_GPUs,
    //                     mype_node, nodesPerPE, numNodes);
    // dense_hidden_forward_uvm(dp5);

    // //layer-5
    // SAG_host_UVM_updated(dsp_out_4, d_den_out_5, 
    //                     d_row_ptr[mype_node], d_col_ind[mype_node],
    //                     lb_src, ub_src, outdim, num_GPUs,
    //                     mype_node, nodesPerPE, numNodes);
    // dense_hidden_forward_uvm(dp6);

    // //softmax
    // softmax_forward(smx2);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time (ms): %.2f\n", milliseconds);
}

#pragma omp parallel for
for (int mype_node = 0; mype_node < num_GPUs; mype_node++)
{
    #ifdef validate
    if (mype_node == validate)
    {
        bool status = compare_array(hd_ref, d_den_out[mype_node], nodesPerPE*dim);
        if (status)
            printf("PE-%d: validate: True\n", mype_node);
        else
            printf("PE-%d: validate: False\n", mype_node);
    }
    #endif

    // hipFree(hd_ref);
    hipFree(d_den_out[mype_node]);
    hipFree(d_input[mype_node]);    
    hipFree(d_col_ind[mype_node]);
    hipFree(d_row_ptr[mype_node]);
}
    hipFree(d_den_out);
    hipFree(d_input);
    hipFree(d_col_ind);
    hipFree(d_row_ptr);
    // free(h_ref);
    // free(h_input);

    return 0;
}